#include "hip/hip_runtime.h"
//-*-c-*-
#include <hip/hip_math_constants.h>
#include <hiprand/hiprand_kernel.h>

#include "linalg.h"
#include "matrix.h"
#include "rotate.h"
#include "mesh.h"
#include "geometry.h"
#include "photon.h"

__device__ void
fAtomicAdd(float *addr, float data)
{
    while (data)
	data = atomicExch(addr, data+atomicExch(addr, 0.0f));
}

__device__ void
to_diffuse(Photon &p, State &s, Geometry *g, hiprandState &rng, int max_steps)
{
    int steps = 0;
    while (steps < max_steps) {
	steps++;

	int command;

	fill_state(s, p, g);

	if (p.last_hit_triangle == -1)
	    break;

	command = propagate_to_boundary(p, s, rng);

	if (command == BREAK)
	    break;

	if (command == CONTINUE)
	    continue;

	if (s.surface_index != -1) {
	    command = propagate_at_surface(p, s, rng, g);

	    if (p.history & REFLECT_DIFFUSE)
		break;

	    if (command == BREAK)
		break;

	    if (command == CONTINUE)
		continue;
	}

	propagate_at_boundary(p, s, rng);

    } // while (steps < max_steps)

} // to_diffuse

extern "C"
{

__global__ void
update_xyz_lookup(int nthreads, int total_threads, int offset, float3 position,
		  hiprandState *rng_states, float wavelength, float3 xyz,
		  float3 *xyz_lookup1, float3 *xyz_lookup2, int max_steps,
		  Geometry *g)
{
    int kernel_id = blockIdx.x*blockDim.x + threadIdx.x;
    int id = kernel_id + offset;

    if (kernel_id >= nthreads || id >= total_threads)
	return;

    hiprandState rng = rng_states[kernel_id];

    Triangle t = get_triangle(g, id);

    float a = hiprand_uniform(&rng);
    float b = uniform(&rng, 0.0f, (1.0f - a));
    float c = 1.0f - a - b;

    float3 direction = a*t.v0 + b*t.v1 + c*t.v2 - position;
    direction /= norm(direction);

    float distance;
    int triangle_index = intersect_mesh(position, direction, g, distance);

    if (triangle_index != id) {
	rng_states[kernel_id] = rng;
	return;
    }

    float3 v01 = t.v1 - t.v0;
    float3 v12 = t.v2 - t.v1;
    
    float3 surface_normal = normalize(cross(v01,v12));

    float cos_theta = dot(surface_normal,-direction);

    if (cos_theta < 0.0f)
	cos_theta = dot(-surface_normal,-direction);

    Photon p;
    p.position = position;
    p.direction = direction;
    p.wavelength = wavelength;
    p.polarization = uniform_sphere(&rng);
    p.last_hit_triangle = -1;
    p.time = 0;
    p.history = 0;

    State s;
    to_diffuse(p, s, g, rng, max_steps);

    if (p.history & REFLECT_DIFFUSE) {
	if (s.inside_to_outside) {
	    fAtomicAdd(&xyz_lookup1[p.last_hit_triangle].x, cos_theta*xyz.x);
	    fAtomicAdd(&xyz_lookup1[p.last_hit_triangle].y, cos_theta*xyz.y);
	    fAtomicAdd(&xyz_lookup1[p.last_hit_triangle].z, cos_theta*xyz.z);
	}
	else {
	    fAtomicAdd(&xyz_lookup2[p.last_hit_triangle].x, cos_theta*xyz.x);
	    fAtomicAdd(&xyz_lookup2[p.last_hit_triangle].y, cos_theta*xyz.y);
	    fAtomicAdd(&xyz_lookup2[p.last_hit_triangle].z, cos_theta*xyz.z);
	}
    }

    rng_states[kernel_id] = rng;

} // update_xyz_lookup

__global__ void
update_xyz_image(int nthreads, hiprandState *rng_states, float3 *positions,
		 float3 *directions, float wavelength, float3 xyz,
		 float3 *xyz_lookup1, float3 *xyz_lookup2, float3 *image,
		 int nlookup_calls, int max_steps, Geometry *g)
{
    int id = blockIdx.x*blockDim.x + threadIdx.x;

    if (id >= nthreads)
	return;

    hiprandState rng = rng_states[id];

    Photon p;
    p.position = positions[id];
    p.direction = directions[id];
    p.direction /= norm(p.direction);
    p.wavelength = wavelength;
    p.polarization = uniform_sphere(&rng);
    p.last_hit_triangle = -1;
    p.time = 0;
    p.history = 0;

    State s;
    to_diffuse(p, s, g, rng, max_steps);

    if (p.history & REFLECT_DIFFUSE) {
	if (s.inside_to_outside)
	    image[id] += xyz*xyz_lookup1[p.last_hit_triangle]/nlookup_calls;
	else
	    image[id] += xyz*xyz_lookup2[p.last_hit_triangle]/nlookup_calls;
    }

    rng_states[id] = rng;

} // update_xyz_image

__global__ void
process_image(int nthreads, float3 *image, unsigned int *pixels, int nimages)
{
    int id = blockIdx.x*blockDim.x + threadIdx.x;

    if (id >= nthreads)
	return;

    float3 rgb = image[id]/nimages;

    if (rgb.x < 0.0f)
	rgb.x = 0.0f;
    if (rgb.y < 0.0f)
	rgb.y = 0.0f;
    if (rgb.z < 0.0f)
	rgb.z = 0.0f;

    if (rgb.x > 1.0f)
	rgb.x = 1.0f;
    if (rgb.y > 1.0f)
	rgb.y = 1.0f;
    if (rgb.z > 1.0f)
	rgb.z = 1.0f;

    unsigned int r = floorf(rgb.x*255.0f);
    unsigned int g = floorf(rgb.y*255.0f);
    unsigned int b = floorf(rgb.z*255.0f);

    pixels[id] = 255 << 24 | r << 16 | g << 8 | b;

} // process_image

} // extern "c"
