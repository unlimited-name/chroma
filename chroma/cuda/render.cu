#include "hip/hip_runtime.h"
//-*-c-*-

#include "linalg.h"
#include "intersect.h"
#include "mesh.h"
#include "sorting.h"
#include "geometry.h"

#include "stdio.h"

__device__ float4
get_color(const float3 &direction, const Triangle &t, unsigned int rgba)
{
    float3 v01 = t.v1 - t.v0;
    float3 v12 = t.v2 - t.v1;
    
    float3 surface_normal = normalize(cross(v01,v12));

    float cos_theta = dot(surface_normal,-direction);

    if (cos_theta < 0.0f)
	cos_theta = -cos_theta;

    unsigned int a0 = 0xff & (rgba >> 24);
    unsigned int r0 = 0xff & (rgba >> 16);
    unsigned int g0 = 0xff & (rgba >> 8);
    unsigned int b0 = 0xff & rgba;

    float alpha = (255 - a0)/255.0f;

    return make_float4(r0*cos_theta, g0*cos_theta, b0*cos_theta, alpha);
}

extern "C"
{

__global__ void
render(int nthreads, float3 *_origin, float3 *_direction, Geometry *g,
       unsigned int alpha_depth, unsigned int *pixels, float *_dx,
       unsigned int *dxlen, float4 *_color, unsigned int bg_color)
{
    __shared__ Geometry sg;

    if (threadIdx.x == 0)
	sg = *g;

    __syncthreads();

    int id = blockIdx.x*blockDim.x + threadIdx.x;
	
    if (id >= nthreads)
	return;
    
    g = &sg;

    float3 origin = _origin[id];
    float3 direction = _direction[id];
    unsigned int n = dxlen[id];

    float distance;

    Node root = get_node(g, 0);

    float3 neg_origin_inv_dir = -origin / direction;
    float3 inv_dir = 1.0f / direction;

    if (n < 1 && !intersect_node(neg_origin_inv_dir, inv_dir, g, root)) {
	pixels[id] = bg_color;
	return;
    }

    unsigned int child_ptr_stack[STACK_SIZE];
    unsigned int nchild_ptr_stack[STACK_SIZE];
    child_ptr_stack[0] = root.child;
    nchild_ptr_stack[0] = root.nchild;

    int curr = 0;

    unsigned int count = 0;
    unsigned int tri_count = 0;

    float *dx = _dx + id*alpha_depth;
    float4 *color_a = _color + id*alpha_depth;

    while (curr >= 0) {
	unsigned int first_child = child_ptr_stack[curr];
	unsigned int nchild = nchild_ptr_stack[curr];
	curr--;

	for (unsigned int i=first_child; i < first_child + nchild; i++) {
	    Node node = get_node(g, i);
	    count++;

	    if (intersect_node(neg_origin_inv_dir, inv_dir, g, node)) {

	      if (node.nchild == 0) { /* leaf node */

		// This node wraps a triangle
		tri_count++;
		Triangle t = get_triangle(g, node.child);
		if (intersect_triangle(origin, direction, t, distance)) {
		  if (n < 1) {
		    dx[0] = distance;
		    
		    unsigned int rgba = g->colors[node.child];
		    float4 color = get_color(direction, t, rgba);
		    
		    color_a[0] = color;
		  }
		  else {
		    unsigned long j = searchsorted(n, dx, distance);
		    
		    if (j <= alpha_depth-1) {
		      insert(alpha_depth, dx, j, distance);
		      
		      unsigned int rgba = g->colors[node.child];
		      float4 color = get_color(direction, t, rgba);
		      
		      insert(alpha_depth, color_a, j, color);
		    }
		  }
		  
		  if (n < alpha_depth)
		    n++;
		  
		} // if hit triangle
		
	      } else {
		curr++;
		child_ptr_stack[curr] = node.child;
		nchild_ptr_stack[curr] = node.nchild;
	      } // leaf or internal node?
	    } // hit node?
	    
	    //if (curr >= STACK_SIZE) {
	    //	printf("warning: intersect_mesh() aborted; node > tail\n");
	    //	break;
	    //}
	} // loop over children, starting with first_child
	
    } // while nodes on stack
    

    if (n < 1) {
	pixels[id] = bg_color;
	return;
    }

    dxlen[id] = n;

    float scale = 1.0f;
    float fr = 0.0;
    float fg = 0.0;
    float fb = 0.0;
    for (int i=0; i < n; i++) {
	float alpha = color_a[i].w;
	
	fr += scale*color_a[i].x*alpha;
	fg += scale*color_a[i].y*alpha;
	fb += scale*color_a[i].z*alpha;
	
	scale *= (1.0f-alpha);
    }
    float alpha = ((float)((bg_color & 0xFF000000) >> 24))/255.0;
    fr += scale*((float)((bg_color & 0xFF0000) >> 16))*alpha;
    fg += scale*((float)((bg_color & 0xFF00) >> 8))*alpha;
    fb += scale*((float)(bg_color & 0xFF))*alpha;
    scale *= (1.0f-alpha);
    
    unsigned int a;
    if (n < alpha_depth)
	a = floorf(255*(1.0f-scale));
    else
    	a = 255;
    	
    unsigned int red = floorf(fr/(1.0f-scale));
    unsigned int green = floorf(fg/(1.0f-scale));
    unsigned int blue = floorf(fb/(1.0f-scale));

    pixels[id] = a << 24 | red << 16 | green << 8 | blue;
}

} // extern "C"
