#include "hip/hip_runtime.h"
//-*-c-*-

#include "matrix.h"

__device__ Matrix array2matrix(float *a)
{
	return make_matrix(a[0], a[1], a[2],
			   a[3], a[4], a[5],
			   a[6], a[7], a[8]);
}

__device__ void matrix2array(const Matrix &m, float *a)
{
	a[0] = m.a00;
	a[1] = m.a01;
	a[2] = m.a02;
	a[3] = m.a10;
	a[4] = m.a11;
	a[5] = m.a12;
	a[6] = m.a20;
	a[7] = m.a21;
	a[8] = m.a22;
}

extern "C"
{

__global__ void det(float *a, float *dest)
{
	Matrix m = array2matrix(a);
	dest[0] = det(m);
}

__global__ void inv(float *a, float *dest)
{
	Matrix m = array2matrix(a);
	matrix2array(inv(m), dest);
}

__global__ void minusmatrix(float *a, float *dest)
{
	matrix2array(-array2matrix(a), dest);
}

__global__ void matrixadd(float *a, float *b, float *dest)
{
	matrix2array(array2matrix(a)+array2matrix(b), dest);
}

__global__ void matrixsub(float *a, float *b, float *dest)
{
	matrix2array(array2matrix(a)-array2matrix(b), dest);
}

__global__ void matrixmul(float *a, float *b, float *dest)
{
	matrix2array(array2matrix(a)*array2matrix(b), dest);
}

__global__ void multiply(float *a, float3 *x, float3 *dest)
{
	dest[0] = array2matrix(a)*x[0];
}

__global__ void matrixaddfloat(float *a, float c, float *dest)
{
	matrix2array(array2matrix(a)+c, dest);
}

__global__ void matrixsubfloat(float *a, float c, float *dest)
{
	matrix2array(array2matrix(a)-c, dest);
}

__global__ void matrixmulfloat(float *a, float c, float *dest)
{
	matrix2array(array2matrix(a)*c, dest);
}

__global__ void matrixdivfloat(float *a, float c, float *dest)
{
	matrix2array(array2matrix(a)/c, dest);
}

__global__ void floataddmatrix(float *a, float c, float *dest)
{
	matrix2array(c+array2matrix(a), dest);
}

__global__ void floatsubmatrix(float *a, float c, float *dest)
{
	matrix2array(c-array2matrix(a), dest);
}

__global__ void floatmulmatrix(float *a, float c, float *dest)
{
	matrix2array(c*array2matrix(a), dest);
}

__global__ void floatdivmatrix(float *a, float c, float *dest)
{
	matrix2array(c/array2matrix(a), dest);
}

__global__ void matrixaddequals(float *a, float *b)
{
	Matrix m = array2matrix(a);
	m += array2matrix(b);
	matrix2array(m,a);
}

__global__ void matrixsubequals(float *a, float *b)
{
	Matrix m = array2matrix(a);
	m -= array2matrix(b);
	matrix2array(m,a);
}

__global__ void matrixaddequalsfloat(float *a, float c)
{
	Matrix m = array2matrix(a);
	m += c;
	matrix2array(m,a);
}

__global__ void matrixsubequalsfloat(float *a, float c)
{
	Matrix m = array2matrix(a);
	m -= c;
	matrix2array(m,a);
}

__global__ void matrixmulequalsfloat(float *a, float c)
{
	Matrix m = array2matrix(a);
	m *= c;
	matrix2array(m,a);
}

__global__ void matrixdivequalsfloat(float *a, float c)
{
	Matrix m = array2matrix(a);
	m /= c;
	matrix2array(m,a);
}

__global__ void outer(float3 a, float3 b, float* dest)
{
	matrix2array(outer(a,b), dest);
}

} // extern "c"
